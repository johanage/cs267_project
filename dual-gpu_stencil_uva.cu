#include "hip/hip_runtime.h"
// This is a 2D stencil example for multi-GPU using CUDA and only UVA

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void stencil_kernel(float *grid, int width, int height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i > 0 && i < width - 1 && j > 0 && j < height - 1)
    {
        // Compute the new value of the grid point (i, j)
        float new_value = (grid[(i-1)*height+j] + grid[(i+1)*height+j]
                            + grid[i*height+j-1] + grid[i*height+j+1]);
        // Write the new value back to the grid
        grid[i*height+j] += new_value;
    }
}

int main() {
    const int width = 16;
    const int height = 16;
    const int num_iterations = 1;
    const int block_size = 4;
    
    // Allocate memory for the grid on both GPUs using hipMallocManaged()
    float *grid1, *grid2;
    hipMallocManaged(&grid1, width*height*sizeof(float));
    hipMallocManaged(&grid2, width*height*sizeof(float));
    
    // Initialize the grid values on GPU 1
    //printf(" Init grid1: \n");
    int sum = 0;
    for (int i = 0; i < int( width*height ); i++) 
    {
        grid1[i] = 1;
	sum += grid1[i];
	std::cout << sum << " ";
    }
    
    // Launch the kernel on both GPUs
    // threads per block
    dim3 block(block_size, block_size);
    int nx = (int)ceil(width/block.x);
    int ny = (int)ceil(height/block.y);
    dim3 grid1_size( nx, ny );
    dim3 grid2_size( nx, ny );

    // prefetching to device
    int device = -1;
    hipGetDevice(&device);
    hipMemPrefetchAsync(grid1, width*height*sizeof(int), device, NULL);
    hipMemPrefetchAsync(grid2, width*height*sizeof(int), device, NULL);

    for (int iter = 0; iter < num_iterations; ++iter) {
        stencil_kernel<<<grid1_size, block>>>(grid1, width, height);
        stencil_kernel<<<grid2_size, block>>>(grid2, width, height);
        hipDeviceSynchronize();
        // Swap the grids so that the updated values are on the other GPU for the next iteration
        float *temp = grid1;
        grid1 = grid2;
        grid2 = temp;
    }
    // copy from GPU to print results on CPU
    int size_grid = int(width*height); 
    printf("Size of grid %i ", size_grid);
    float *grid   = new float[size_grid]();
    float *grid_2 = new float[size_grid]();
    hipMemcpy(grid,   grid1, size_grid*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(grid_2, grid2, size_grid*sizeof(int), hipMemcpyDeviceToHost);
    // Free the memory using hipFree()
    hipFree(grid1);
    hipFree(grid2);
    printf("Printing the output of the 2D stencil example\n");
    for(int i = 0; i < width; i++)
    {
        for(int j = 0; j < height; j++)
	{
		//std::cout << i*width + j << " : " << grid_2[i*width + j] << " "; // << std::endl;
		//std::cout << grid[i*width + j] << " "; // << std::endl;
		std::cout << grid_2[i*width + j] << " "; // << std::endl;
		//std::cout << grid[i*width + j] + grid_2[i*width + j] << " "; // << std::endl;
	}
	std::cout << std::endl;
    }
    return 0;
}

