#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "nccl.h"

#define NX 1024
#define NY 1024
#define NZ 1024
#define HALO 1

__global__ void stencil(float *in, float *out, int nx, int ny, int nz)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int k = threadIdx.z + blockIdx.z * blockDim.z;

    if (i >= HALO && i < nx - HALO && j >= HALO && j < ny - HALO && k >= HALO && k < nz - HALO) {
        out[i * ny * nz + j * nz + k] = (in[(i - 1) * ny * nz + j * nz + k] +
                                         in[(i + 1) * ny * nz + j * nz + k] +
                                         in[i * ny * nz + (j - 1) * nz + k] +
                                         in[i * ny * nz + (j + 1) * nz + k] +
                                         in[i * ny * nz + j * nz + k - 1] +
                                         in[i * ny * nz + j * nz + k + 1]) / 6.0f;
    }
}

int main(int argc, char *argv[])
{
    int nGPUs = 2;
    int deviceIDs[2] = {0, 1};

    // Initialize NCCL
    ncclComm_t comm;
    ncclUniqueId id;
    ncclGetUniqueId(&id);
    ncclCommInitRank(&comm, nGPUs, id, 0, deviceIDs);

    // Allocate memory on each GPU
    hipSetDevice(deviceIDs[0]);
    float *d_in0, *d_out0;
    hipMalloc((void **)&d_in0, NX * NY * NZ * sizeof(float));
    hipMalloc((void **)&d_out0, NX * NY * NZ * sizeof(float));

    hipSetDevice(deviceIDs[1]);
    float *d_in1, *d_out1;
    hipMalloc((void **)&d_in1, NX * NY * NZ * sizeof(float));
    hipMalloc((void **)&d_out1, NX * NY * NZ * sizeof(float));

    // Initialize input data on the first GPU
    hipSetDevice(deviceIDs[0]);
    float *h_in = (float *)malloc(NX * NY * NZ * sizeof(float));
    for (int i = 0; i < NX * NY * NZ; i++) {
        h_in[i] = i;
    }
    hipMemcpy(d_in0, h_in, NX * NY * NZ * sizeof(float), hipMemcpyHostToDevice);

    // Synchronize NCCL
    ncclCommCuDevice(comm, deviceIDs[0]);
    hipStream_t stream0;
    hipStreamCreate(&stream0);
    ncclGroupStart();
    ncclSend(d_in0, NX * NY * NZ, ncclFloat, 0, comm, stream0);
    hipSetDevice(deviceIDs[1]);
    float *d_in1_recv;
    hipMalloc((void **)&d_in1_recv, NX * NY * NZ * sizeof(float));
    hipStream_t stream1;
    hipStreamCreate(&stream1);
    ncclRecv(d_in1, NX * NY * NZ, ncclFloat, 0, comm, stream1);
	ncclGroupEnd();
	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);
	// Perform the stencil operation on both GPUs
	dim3 blockDim(8, 8, 8);
	dim3 gridDim((NX + blockDim.x - 1) / blockDim.x, (NY + blockDim.y - 1) / blockDim.y, (NZ + blockDim.z - 1) / blockDim.z);
	hipSetDevice(deviceIDs[0]);
	stencil<<<gridDim, blockDim>>>(d_in0, d_out0, NX, NY, NZ);

	hipSetDevice(deviceIDs[1]);
	stencil<<<gridDim, blockDim>>>(d_in1_recv, d_out1, NX, NY, NZ);

	// Synchronize NCCL
	ncclCommCuDevice(comm, deviceIDs[1]);
	hipStream_t stream2;
	hipStreamCreate(&stream2);
	ncclGroupStart();
	ncclSend(d_out1, NX * NY * NZ, ncclFloat, 1, comm, stream2);
	hipSetDevice(deviceIDs[0]);
	float *d_out0_recv;
	hipMalloc((void **)&d_out0_recv, NX * NY * NZ * sizeof(float));
	hipStream_t stream3;
	hipStreamCreate(&stream3);
	ncclRecv(d_out0_recv, NX * NY * NZ, ncclFloat, 1, comm, stream3);
	ncclGroupEnd();
	hipStreamSynchronize(stream2);
	hipStreamSynchronize(stream3);

	// Combine the results on the first GPU
	hipSetDevice(deviceIDs[0]);
	for (int i = HALO; i < NX - HALO; i++) {
		for (int j = HALO; j < NY - HALO; j++) {
			for (int k = HALO; k < NZ - HALO; k++) {
				h_in[i * NY * NZ + j * NZ + k] = (d_out0_recv[i * NY * NZ + j * NZ + k] + d_out0[i * NY * NZ + j * NZ + k]) / 2.0f;
			}
		}
	}

	// Free memory
	hipFree(d_in0);
	hipFree(d_out0);
	hipFree(d_in1);
	hipFree(d_out1);
	hipFree(d_in1_recv);
	hipFree(d_out0_recv);
	free(h_in);

	// Finalize NCCL
	ncclCommDestroy(comm);

	return 0;
}
