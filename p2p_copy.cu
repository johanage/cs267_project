#include <hip/hip_runtime.h>

#include <stdio.h>

float p2p_copy (size_t size)
{
	int *pointers[2];

	hipSetDevice (0);
	hipDeviceEnablePeerAccess (1, 0);
	hipMalloc (&pointers[0], size);

	hipSetDevice (1);
	hipDeviceEnablePeerAccess (0, 0);
	hipMalloc (&pointers[1], size);

	hipEvent_t begin, end;
	hipEventCreate (&begin);
	hipEventCreate (&end);

	hipEventRecord (begin);
	hipMemcpyAsync (pointers[0], pointers[1], size, hipMemcpyDeviceToDevice);
	hipEventRecord (end);
	hipEventSynchronize (end);

	float elapsed;
	hipEventElapsedTime (&elapsed, begin, end);
	elapsed /= 1000;

	hipSetDevice (0);
	hipFree (pointers[0]);

	hipSetDevice (1);
	hipFree (pointers[1]);

	hipEventDestroy (end);
	hipEventDestroy (begin);

	return elapsed;
}

int main(){
	float elapse_p2p_copy;
	size_t size = 100000;
	elapse_p2p_copy = p2p_copy(size);
	printf("Elapsed time p2p copy: %f s \n", elapse_p2p_copy);
}
