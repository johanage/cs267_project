#include <hip/hip_runtime.h>
#include <iostream>
void print_ptr_dev (int max_depth, int current_depth = 0)
{
  hipSetDevice (current_depth % 2);

  void *ptr {};

  hipMalloc (&ptr, 1024 * 1024 * 4);

  hipPointerAttribute_t pointer_attributes {};
  hipPointerGetAttributes (&pointer_attributes, ptr);

  std::cout << (size_t)ptr << ", " << pointer_attributes.device << "\n";

  if (max_depth > current_depth)
      print_ptr_dev (max_depth, current_depth + 1);

  hipFree (ptr);
}

int main ()
{
  print_ptr_dev (40);
  return 0;
}
